#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "triangle.h"
#include "helpers.h"

#include "vec_math.h"

extern "C" {
__constant__ Params params;
}


static __forceinline__ __device__ void setPayload( float p, unsigned int i)
{
    optixSetPayload_0( float_as_int(p) );
    optixSetPayload_1(i);
}


extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    // Map our launch idx to a screen location and create a ray from the camera
    // location through the screen
    float3 ray_origin, ray_direction;
    ray_origin = params.rays_o[idx.y * params.image_width + idx.x];
    ray_direction = params.rays_d[idx.y * params.image_width + idx.x];

    // Trace the ray against our scene hierarchy
    float result = 0.0f;
    unsigned int p0;
    unsigned int p1;
    optixTrace(
            params.handle,
            ray_origin,
            ray_direction,
            0.0f,                // Min intersection distance
            1e16f,               // Max intersection distance
            0.0f,                // rayTime -- used for motion blur
            OptixVisibilityMask( 255 ), // Specify always visible
            OPTIX_RAY_FLAG_NONE,
            0,                   // SBT offset   -- See SBT discussion
            1,                   // SBT stride   -- See SBT discussion
            0,                   // missSBTIndex -- See SBT discussion
            p0, p1 );
    result = int_as_float( p0 );

    // Record results in our output raster
    params.hit[idx.y * params.image_width + idx.x] = result;
    params.triangle_index[idx.y * params.image_width + idx.x] = p1;
}


extern "C" __global__ void __miss__ms()
{
    MissData* miss_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    setPayload( 0.0f , 10000000u);
}


extern "C" __global__ void __closesthit__ch()
{
    // When built-in triangle intersection is used, a number of fundamental
    // attributes are provided by the OptiX API, indlucing barycentric coordinates.
    const float2 barycentrics = optixGetTriangleBarycentrics();

    setPayload( 1.0f,  optixGetPrimitiveIndex() );
}
